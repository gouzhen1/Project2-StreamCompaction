#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <iostream>
#include <cmath>
#include "CPU_streamCompaction.h"

using namespace std;
#define BLOCKDIM 128;

float Log2(float n)
{
	return log(n)/log(2);
}

__global__ void dev_initialize_array(int n, float * tar, float val)
{
	int index = (blockIdx.x * blockDim.x) + threadIdx.x;
	if(index < n) tar[index] = val;
}

__global__ void NaiveInclusivePrefixSum(int D, float * input, float * buffer, int n)
{
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	//copy input to buffer
	buffer[index] = input[index];
	__syncthreads();
	if(index < n && index >= D)
	{
		input[index] = buffer[index - D ] + buffer[index ];
	}
}
__global__ void NaiveExclusivePrefixSum(int D, float * input,float * output, float * buffer, int n)
{
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	//copy input to buffer
	buffer[index] = input[index];
	__syncthreads();
	if(index < n && index >= D)
	{
		input[index] = buffer[index - D ] + buffer[index ];
	}

	__syncthreads();
	output[index] = (index>0) ? input[index-1]:0.0f;
}

__global__ void AddAuxToBlockedPrefixSum(float * input, float * aux, float * res, int n)
{
	int index = blockDim.x * blockIdx.x + threadIdx.x;
	if(index < n && blockIdx.x > 0) input[index] += aux[blockIdx.x - 1];
	if(index < n+1)	res[index] = (index <1)? 0.0f : input[index - 1];

}

__global__ void SingleBlockExclusivePrefixSum(int D, float * input,float * output, int n)
{
	extern __shared__ float buffer[];	
	int index = threadIdx.x;
	//copy input to buffer
	if(index < n) buffer[index] = input[index];
	__syncthreads();

	if(index < n && index >= D)
	{
		input[index] = buffer[index - D ] + buffer[index];
	}

	__syncthreads();
	output[index] = (index>0) ? input[index-1]:0.0f;
}

__global__ void MultiBlockInclusivePrefixSum(int D, float * input, float * aux, int n)
{
	extern __shared__ float buffer[];	
	int localIndex = threadIdx.x;
	int globalIndex = blockDim.x * blockIdx.x + localIndex;
	//copy input to buffer
	if(localIndex < n) buffer[localIndex] = input[globalIndex];
	__syncthreads();

	if(localIndex < n && localIndex >= D)
	{
		input[globalIndex] = buffer[localIndex - D ] + buffer[localIndex];
	}

	__syncthreads();
	//if( globalIndex < n) output[globalIndex] = input[globalIndex];
	if(localIndex ==  blockDim.x - 1) aux[blockIdx.x] = (globalIndex>n)? input[n-1]: input[globalIndex];
}

void NaiveGPUexclusiveScan(float * input, float * output, int n)
{
	int blockSize = BLOCKDIM;
	int gridSize = ceil((float)n/(float)blockSize); 
	float * dev_buffer;
	hipMalloc((void**) & dev_buffer, (n + 1) * sizeof(float));

	dim3 gridDim(gridSize);
	dim3 blockDim(blockSize);
	for(int i=1;i<=Log2(n) + 1;i++)
	{
		int D = pow(2,i-1);
		NaiveExclusivePrefixSum<<<gridDim,blockDim>>>(D,input,output, dev_buffer,n);
	}

}

void GPUexclusiveScan(float * input, float * output, int n)
{
	int blockDim = BLOCKDIM;
	int gridDim = ceil((float)n/(float)blockDim);
	float *dev_input, * dev_aux, * dev_buffer;
	hipMalloc((void**) & dev_input, (n) * sizeof(float));
	hipMalloc((void**) & dev_aux, (gridDim) * sizeof(float));
	hipMalloc((void**) & dev_buffer, (n + 1) * sizeof(float));

	hipMemcpy(dev_input, input, n*sizeof(float),hipMemcpyDeviceToDevice);

	int D(0);
	for(int i=1;i< Log2(n) + 1;i++)
	{
		D = pow(2,i-1);
		MultiBlockInclusivePrefixSum<<<gridDim,blockDim,n*sizeof(float)>>>(D,dev_input,dev_aux,n);
	}
	/*
	//print aux
	hipMemcpy(res, dev_aux, (gridDim)*sizeof(float),hipMemcpyDeviceToHost);
	cout<<"aux: ";
	for(int i=0;i<gridDim;i++)
	{
		cout<<res[i]<<" ";
	}
	cout<<endl;*/
	//scan aux
	for(int i=1;i< Log2(gridDim) + 1;i++)
	{
		D = pow(2,i-1);
		NaiveInclusivePrefixSum<<<ceil((float)gridDim/(float)blockDim),blockDim,gridDim*sizeof(float)>>>(D,dev_aux,dev_buffer,gridDim);
	}
	//print scanned aux
	/*
	hipMemcpy(res, dev_aux, (gridDim)*sizeof(float),hipMemcpyDeviceToHost);
	cout<<"scanned aux: ";
	for(int i=0;i<gridDim;i++)
	{
		cout<<res[i]<<" ";
	}
	cout<<endl;*/

	//add aux to dev_in
	AddAuxToBlockedPrefixSum<<<gridDim+1,blockDim>>>(dev_input,dev_aux,output,n+1);
}

__global__ void generateBoolArray(float * input, float * out, int n)
{
	int index = blockDim.x * blockIdx.x + threadIdx.x;
	if(index <n) out[index] = (input[index] == 0.0f )? 0.0f: 1.0f;
}

__global__ void generateCompactArray(float * input,float * boolArray, float * scannedBool, float * output, int n)
{
	int index = blockDim.x * blockIdx.x + threadIdx.x;
	if(index < n)
	{
		if(boolArray[index] > 0.0f) output[(int)scannedBool[index]] = input[index];
	}
}
void GPUstreamCompaction(float * input, float * output, int n)
{
	int blockDim = BLOCKDIM;
	int gridDim = ceil((float)n/(float)blockDim);

	float * boolArray, * scannedBool;
	hipMalloc((void**) & boolArray, n * sizeof(float));
	hipMalloc((void**) & scannedBool, (n+1) * sizeof(float));

	generateBoolArray<<<gridDim,blockDim>>>(input,boolArray,n);
	GPUexclusiveScan(boolArray,scannedBool,n);
	generateCompactArray<<<gridDim,blockDim>>>(input,boolArray, scannedBool, output, n);

}

int main(int argc, char** argv)
{
	//timers
	hipEvent_t start, stop; 
	float time = 0.0f;
	//init
	float * in, *res, *dev_in, * dev_res;
	int n = 1000000;
	in = (float*)malloc(n * sizeof(float));
	res = (float*)malloc((1+n) * sizeof(float));
	hipMalloc((void**) & dev_in, n * sizeof(float));
	hipMalloc((void**) & dev_res, (n+1) * sizeof(float));
	//load data
	for(int i=0;i<n;i++)
	{
		//in[i] = (float) i;
		in[i] = (i%2 == 0) ? i : 0.0f;
	}
//	in[0] = 1.0f;in[1] = 3.0f;in[2] = 2.0f;in[3] = 1.0f;in[4] = 4.0f;in[5] = 2.5f;
	hipMemcpy(dev_in,in,n * sizeof(float),hipMemcpyHostToDevice);

	//print input
	cout<<"input: ";
	for(int i=0;i<n;i++)
	{
		//cout<<in[i]<<" ";
	}
	cout<<endl;

	//CPU exprefixsum////////////////////////////////////////////////////////////////////////////////
	
	CPUstreamCompaction(in,n,res);

	cout<<"CPU stream compact runtime: "<<time<<" ms"<<endl;
	cout<<"CPU compact stream result: ";
	for(int i=0;i<n+1;i++)
	{
		//cout<<res[i]<<" ";
	}
	cout<<endl;
	///////////////////////////////////////////////////////////////////////////////////////////////////////

	#if(1)//naive GPU ex prefix sum//////////////////////////////////////////////////////////////////
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord( start, 0 );
	NaiveGPUexclusiveScan(dev_in,dev_res,n);
	hipEventRecord( stop, 0 );
	hipEventSynchronize( stop );
	hipEventElapsedTime( &time, start, stop );
	hipEventDestroy( start );
	hipEventDestroy( stop );
	cout<<"Naive GPU scan runtime: "<<time<<" ms"<<endl;
	#endif////////////////////////////////////////////////////////////////////////////////////////////////

	#if(1)//naive GPU ex prefix sum//////////////////////////////////////////////////////////////////
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord( start, 0 );
	GPUexclusiveScan(dev_in,dev_res,n);
	hipEventRecord( stop, 0 );
	hipEventSynchronize( stop );
	hipEventElapsedTime( &time, start, stop );
	hipEventDestroy( start );
	hipEventDestroy( stop );
	cout<<"GPU scan runtime: "<<time<<" ms"<<endl;
	#endif////////////////////////////////////////////////////////////////////////////////////////////////

	#if(0)//single block with shared memory ex prefix sum////////////////////////////////////////////////////////
	for(int i=1;i< Log2(n) + 1;i++)
	{
		int D = pow(2,i-1);
		SingleBlockExclusivePrefixSum<<<1,n+1,n*sizeof(float)>>>(D,dev_in,dev_res,n);
	}
	#endif////////////////////////////////////////////////////////////////////////////////////////////


	#if(0)//////////////////////////////////////////////////////////////////////////////////////////////////////
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord( start, 0 );

	GPUstreamCompaction(dev_in,dev_res,n);

	hipEventRecord( stop, 0 );
	hipEventSynchronize( stop );
	hipEventElapsedTime( &time, start, stop );
	hipEventDestroy( start );
	hipEventDestroy( stop );


	cout<<"GPU stream compact runtime: "<<time<<" ms"<<endl;
	#endif//////////////////////////////////////////////////////////////////////////////////////////////////////

	hipMemcpy(res, dev_res, (n)*sizeof(float),hipMemcpyDeviceToHost);
	cout<<"GPU stream compact result: ";
	for(int i=0;i<n+1;i++)
	{
		//cout<<res[i]<<" ";
	}

	cin.get();
    return 0;
}
